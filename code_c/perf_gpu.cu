/* Teste la peformance de rsqrt sur un grand nombre de valeurs aléatoires (version GPU)
 * À compiler avec `nvcc perf_gpu.cu -o test -O3` (requière CUDA!)
 */


#include <cmath>
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

#define N_FLOAT 100000000
#define MAX_FLOAT 1000

__global__ void rsqrt_vec(float* vec_source, int n)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) // évite les débordements
        vec_source[i] = rsqrtf(vec_source[i]);
}

int main() {
    float *floats_source = NULL, *floats_dest = NULL;
    float *d_comm = NULL;
    int i;
    
    hipSetDevice(0);
    srand((unsigned) time(NULL));
    
    // créee le tableau de flottants
    floats_source = (float*) malloc(N_FLOAT *sizeof(float));
    
    if(floats_source == NULL) {
        printf("error allocating floats_source");
        return -1;
    }
    
    for(i=0; i < N_FLOAT; i++) // génère N_FLOAT float entre 1 et MAX_FLOAT
        floats_source[i] = ((float) (1 + rand())) / RAND_MAX * MAX_FLOAT;
    
    // alloue de la mémoire pour le résultat
    floats_dest = (float*) malloc(N_FLOAT *sizeof(float));
    
    if(floats_dest == NULL) {
        printf("error allocating floats_dest");
        return -1;
    }
    
    // crée la mémoire sur le GPU
    hipMalloc(&d_comm, N_FLOAT *sizeof(float));
    
    // teste les performances:
    auto before = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_comm, floats_source, N_FLOAT *sizeof(float), hipMemcpyHostToDevice);
    
    int blocksize = 512;
    int nblock = N_FLOAT/blocksize + (N_FLOAT % blocksize > 0 ? 1: 0);
    rsqrt_vec<<<nblock, blocksize>>>(d_comm, N_FLOAT);
    
    hipMemcpy(floats_dest, d_comm, N_FLOAT *sizeof(float), hipMemcpyDeviceToHost);
    auto t_rsqrt = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - before);
    
    std::cout << "Le tout a prit " 
              << ((double) t_rsqrt.count() / 1e6)
              << "ms, donc rsqrt: " 
              << ((double) t_rsqrt.count() / N_FLOAT) 
              << " ns/floats" << std::endl;
    
    free(floats_source);
    free(floats_dest);
    hipFree(d_comm);
}
